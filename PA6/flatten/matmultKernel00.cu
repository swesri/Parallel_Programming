
#include <hip/hip_runtime.h>

//type0:MM,type1:MTM,type2:MMT
#define by 2
#define bx 2
__global__ void MatMulKernel(double* A, double* B, double* C, long N, long M, long P, int type){

 
  double *Asub, *Bsub, *Csub;

  int thread_col = threadIdx.y;
  int thread_row = threadIdx.x;
  int block_col = blockIdx.y;
  int block_row = blockIdx.x;

  Csub = &C[P * blockDim.y * block_row + blockDim.x * block_col];

  float Cvalue = 0;

if(type==0)
{
  for (int m = 0;  m < (M / blockDim.x); ++m){
    
    Asub = &A[M * blockDim.y * block_row + blockDim.x * m];
    Bsub = &B[M * blockDim.x * block_col + blockDim.y * m];


    __shared__ float shared_A[by][bx];
    __shared__ float shared_B[bx][by];

  
    shared_A[thread_row][thread_col] = Asub[thread_row * M/*blockDim.x*/ + thread_col];
    shared_B[thread_row][thread_col] = Bsub[thread_row * P/*blockDim.y*/ + thread_col];

   
    __syncthreads();

    
#pragma unroll
    for(int e=0; e<blockDim.x; ++e)
       Cvalue += shared_A[thread_row][e] * shared_B[e][thread_col];

    __syncthreads();
  }
}

printf("(%d,%d)=%d\n",blockIdx.x,blockIdx.y,threadIdx.x*P+threadIdx.y);
  Csub[thread_row * P/*blockDim.x*/ + thread_col] = Cvalue;
}

