#include "hip/hip_runtime.h"
///
/// matmultKernel00.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-01-27
/// Last Modified: 2011-02-23 DVN
///
/// Multiplies two matrices using CUDA: A x B = C
///
/// Copy this file and modify the MatMultKernel device function for
/// each of your experiments. 
///

#include "matmultKernel.h"
#include<stdio.h>
#define FOOTPRINT_SIZE BLOCK_SIZE

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

  // matrix blocks
  float *Asub, *Bsub, *Csub;
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Each THREAD BLOCK computes one sub matrix Csub of C
  // EACH THREAD creates its own matrix descriptor Csub
  Csub = &C.elements[C.stride * BLOCK_SIZE * block_row + BLOCK_SIZE * block_col];

  // Each thread computes one element of Csub in its copy of CValue
  float Cvalue = 0;
  float temp[2][2];
  // Loop over all sub matrices in block_row of A and block_col of B
  // required to compute Csub. Block multiply each pair of sub matrices
  // and accumulate results
  for (int m = 0;  m < (A.width / BLOCK_SIZE); ++m){
    // Get Asub and Bsub descriptors
    Asub = &A.elements[A.stride * BLOCK_SIZE * block_row + BLOCK_SIZE * m];
    Bsub = &B.elements[B.stride * BLOCK_SIZE * m + BLOCK_SIZE * block_col];

    // Copy ELEMENTS OF  ASub and Bsub into shared memory
    // EACH THREAD loads ONE ELEMENT of ASub and ONE of Bsub
    // Notice: it does not need to be the element it requires to
    //         compute its Cvalue, as long as all elements are 
    //         collaboratively read. 

    // Notice: every thread declares shared_A and shared_B in shared memory
    //         even though a thread block has only one shared_A and one shared_B
    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    // Each thread copies just one element of shared_A and one element of shared_B
    
        shared_A[thread_row][thread_col] = Asub[thread_row * A.stride + thread_col];
        shared_B[thread_row][thread_col] = Bsub[thread_row * B.stride + thread_col];

        // Synchronize to ensure all elements are read
        __syncthreads();
     
   
     // Do an inproduct of one row of shared_A and one col of shared_B
     // computing one Cvalue by accumulation
   // #pragma unroll
   int i=thread_row;
   int j=thread_col;
   //__shared__ float temp[2][2];
  int k=0;
  int m=0;
  int n,l; 
  // if((threadIdx.x==1)&&(threadIdx.y==1)&&(blockIdx.x==1)&&(blockIdx.y==1)) {
   for(int i=thread_row, n=0;n<=1;n++) {
     for(int j=thread_col,l=0;l<=1;l++) {
    for(int e=0; e<BLOCK_SIZE; ++e){
       Cvalue += shared_A[i][e] * shared_B[e][j];
     
      //printf("\t e=%d: %f",e,shared_A[i][e]);
    }
   //__syncthreads();
//   }
Csub[i*C.stride+j]=Cvalue;//temp[k][m]; 
    j=j+16;
   } //end of j
   i=i+16;
 } //end of i

 __syncthreads();
//}
}
 /*Csub[thread_row*C.stride+thread_col]=temp[0][0];//Cvalue;
 Csub[(thread_row+16)*C.stride+thread_col]=temp[1][0];
 Csub[thread_row*C.stride+(thread_col+16)]=temp[0][1];
 Csub[(thread_row+16)*C.stride+(thread_col+16)]=temp[1][1];*/
  



  // Write Csub to GLOBAL memory.
  // Each thread writes its own cell value.
  
}

