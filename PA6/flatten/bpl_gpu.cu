#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------------------------------------------*/
/// bpl.c
/// For CSU CS475 Fall 2016
/// Instructor: Sanjay Rajopadhye
/// GTA: Swetha Varadarajan
/// Based on code Created by Paul Tero at Existor Ltd as part of a neural networks tutorial
/// Modified by Swetha Varadarajan
/// Created: 2016-11-16
/*---------------------------------------------------------------------------------------------------------------*/

#include <stdio.h> 
#include <stdlib.h> 
#include <string.h> 
#include <math.h> 

#include "timer.h"
#include "util.h"
#include "bunch-new.h"
#include "bplKernel.h"
#include "matmultKernel00.cu"
#define X(i,j) X[((i)*(cmdLineArgs.N+1))+(j)]
#define H(i,j) H[((i)*(cmdLineArgs.M+1))+(j)]

double* d_A; 
double* d_B; 
double* d_C;
double* d_H;
double* d_Zh;
double* d_Zy;
double* d_P;
double* d_X;
double* d_Wxh;


// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

int main(int argc, char** argv) 
{

/*---------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------Command line parsing--------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/

  Params cmdLineArgs;
  parseCmdLineArgs(&cmdLineArgs,argc,argv);

/*---------------------------------------------------------------------------------------------------------------*/
/*-------------------------------------------Variable Declaration------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/

  /*Array description and its size in the comments next to its declation*/

  double *inputs;//Given inputs = total number of samples(S)*number of inputs per sample(N) 
  double *outputs;//Expected outputs = total number of samples(S)*number of outputs per sample(P) 

  double *X;//Input for a given iteration = bunch size(I)*number of inputs per sample(N+1(bias))
  double *Y;//Output for a given iteration = bunch size(I)*number of outputs per sample(P)

  double *Wxh; //Weights in between input and hidden layer = (N+1)*M
  double *Why; //Weights in between input and hidden layer = (M+1)*P
  double *dWxh; //Error Weights in between input and hidden layer = (N+1)*M
  double *dWhy; //Error Weights in between input and hidden layer = (M+1)*P

  double *Zh; //Weighted sum for hidden layer=I*M
  double *H;  // Activation values = I*(M+1)
  double *Zy; //Weighted sum for output layer=I*P 
  double *E;  //Calculated Errors = I*P
  double *P1; //Oredicted output = I*P
  double *P;  // (exp(Zy)) = I*P
  double *sum; //(summation of the P[i]s) = I
  
  double learningrate = 0.0001; /*learning rate */
  long b = cmdLineArgs.sample_per_iter;
  
  long k2 = cmdLineArgs.sample_total/b ; /*number of full bunches */
  long k3 = cmdLineArgs.sample_total-(k2*b); /* size of the partial bunch */

   dim3 dimGrid(b);                    
   dim3 dimBlock(cmdLineArgs.P); 
/*---------------------------------------------------------------------------------------------------------------*/
/*-------------------------------------------Memory allocations--------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
 
  inputs  = (double*)malloc(cmdLineArgs.sample_total * sizeof(double) * cmdLineArgs.N);
  outputs = (double*)malloc(cmdLineArgs.sample_total * sizeof(double) * cmdLineArgs.P);
  
  sum	  = (double*)malloc((b)*sizeof(double));

  /*for(long i = 0; i < cmdLineArgs.sample_total; ++i )
  {
	inputs[i] =(double*)malloc(cmdLineArgs.N * sizeof(double));
	outputs[i]=(double*)malloc(cmdLineArgs.P * sizeof(double));
  }*/

  Wxh     = (double*)malloc((cmdLineArgs.N+1) * sizeof(double) *cmdLineArgs.M);
  Why	  = (double*)malloc((cmdLineArgs.M+1) * sizeof(double) *cmdLineArgs.P);
  dWxh    = (double*)malloc((cmdLineArgs.N+1) * sizeof(double) *cmdLineArgs.M);
  dWhy	  = (double*)malloc((cmdLineArgs.M+1) * sizeof(double) *cmdLineArgs.P);

  /*for(long i = 0; i < cmdLineArgs.N+1; ++i )
  {
	Wxh[i] =(double*)malloc(cmdLineArgs.M * sizeof(double));	
	dWxh[i]=(double*)malloc(cmdLineArgs.M * sizeof(double));
  }

  for(long i = 0; i < cmdLineArgs.M+1; ++i )
  {
	Why[i] =(double*)malloc(cmdLineArgs.P * sizeof(double));
	dWhy[i]=(double*)malloc(cmdLineArgs.P * sizeof(double));
  }*/

  X	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.N+1));
  E	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));
  P	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));
  P1  	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));
  H	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.M+1));
  Zh  	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.M));
  Zy  	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));

  /*for(long i = 0; i < b; ++i )
  {
  X[i]	  = (double*)malloc((cmdLineArgs.N+1)*sizeof(double));
  E[i]	  = (double*)malloc(cmdLineArgs.P*sizeof(double));
  P[i]	  = (double*)malloc(cmdLineArgs.P*sizeof(double));
  P1[i]   = (double*)malloc(cmdLineArgs.P*sizeof(double));
  H[i]	  = (double*)malloc((cmdLineArgs.M+1)*sizeof(double));
  Zh[i]	  = (double*)malloc(cmdLineArgs.M*sizeof(double));
  Zy[i]	  = (double*)malloc(cmdLineArgs.P*sizeof(double));
  }*/

  if( inputs == NULL || outputs == NULL || X == NULL|| H == NULL || dWxh == NULL || dWhy == NULL 
      || Zh == NULL || Zy == NULL || Wxh == NULL || Why == NULL|| E == NULL || P == NULL
	  || P1 == NULL || sum == NULL)
  {
    printf( "Could not allocate memory\n" );
    exit(0);
  }
size_t size = b * cmdLineArgs.P * sizeof(double);

   hipError_t error;
       error = hipMalloc((void**)&d_A, size);
	       if (error != hipSuccess) Cleanup(false);
		       error = hipMalloc((void**)&d_B, size);
			       if (error != hipSuccess) Cleanup(false);
				       error = hipMalloc((void**)&d_C, size);
					       if (error != hipSuccess) Cleanup(false);
error = hipMalloc((void**)&d_H, b * (cmdLineArgs.M+1) * sizeof(double));
           if (error != hipSuccess) Cleanup(false);
		                  error = hipMalloc((void**)&d_Zh, b * cmdLineArgs.M * sizeof(double));
						                     if (error != hipSuccess) Cleanup(false);

error = hipMalloc((void**)&d_P, b * (cmdLineArgs.P) * sizeof(double));
           if (error != hipSuccess) Cleanup(false);
		                             error = hipMalloc((void**)&d_Zy, b * cmdLineArgs.P * sizeof(double));
									                                              if (error != hipSuccess) Cleanup(false);
printf("Hi\n");
error = hipMalloc((void**)&d_X, b*(cmdLineArgs.N+1)*sizeof(double));
if(error!= hipSuccess) Cleanup(false);
error = hipMalloc((void**)&d_Wxh, cmdLineArgs.M*(cmdLineArgs.N+1)*sizeof(double));
if(error!= hipSuccess) Cleanup(false);
//error = hipMalloc((void**)&d_Zh, b*(cmdLineArgs.M)*sizeof(double));
//if(error!= hipSuccess) Cleanup(false);

/*---------------------------------------------------------------------------------------------------------------*/
/*----------------------------------------------Initializations--------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/

  initializeW(Wxh,(cmdLineArgs.N+1),cmdLineArgs.M);
  initializeW(Why,(cmdLineArgs.M+1),cmdLineArgs.P);
  initializeI(inputs,cmdLineArgs.sample_total,cmdLineArgs.N);
  initializeO(outputs,cmdLineArgs.sample_total,cmdLineArgs.P);
//printf("Initialize over\n");
/*---------------------------------------------------------------------------------------------------------------*/
/*------------------------------------------------Training-------------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
  initialize_timer();
  start_timer();
   //displayMatrix1 ("inputs", inputs, cmdLineArgs.sample_total, cmdLineArgs.N);
  for (long t=0; t<cmdLineArgs.iter; t++) //Time loop
  {
//  printf("Time loop:%ld\n",t);
	 for (long s=0; s<k2; s++) //Bunch loop
	  { 	
//	  printf("Bunch loop:%ld\n",s);
		for(long i=0;i<b;i++)
		{
		X(i,0)=H(i,0)=1;//bias setting
		//required input/output are copied from inputs/outputs to X and Y
	 	memcpy (&X(i,1), &inputs[cmdLineArgs.N*((s*b)+i)], cmdLineArgs.N*sizeof(double)); 
		}
		Y = &outputs[s*b*cmdLineArgs.P]; 
		 //displayMatrix1 ("expected input", X, b, cmdLineArgs.N+1);
		  //displayMatrix1 ("input/hidden weights", Wxy, , cmdLineArgs.P);
		/*Forward Phase*/
//		printf("Forward Phase\n");
//		mm(Zh,X,Wxh,b,cmdLineArgs.N+1,cmdLineArgs.M); //Zh=X*Wxh
//printf("Hello0\n");
error=hipMemcpy(d_Wxh,Wxh,(cmdLineArgs.N+1)*cmdLineArgs.M*sizeof(double),hipMemcpyHostToDevice);
if(error!=hipSuccess)Cleanup(false);
error=hipMemcpy(d_X,X,b*(cmdLineArgs.N+1)*sizeof(double),hipMemcpyHostToDevice);
if(error!=hipSuccess)Cleanup(false);
dim3 dimGri1(2,2);
   dim3 dimBlock1(2,2);
//printf("Hello1\n");
MatMulKernel<<<dimGri1,dimBlock1>>>(d_X,d_Wxh,d_Zh,b,cmdLineArgs.N+1,cmdLineArgs.M,0);
func(H,Zh,b,cmdLineArgs.M,1); //H=f1(Zh)
printf("Hello2\n");
error=hipMemcpy(Zh,d_Zh,b*cmdLineArgs.M*sizeof(double),hipMemcpyDeviceToHost);
if(error!=hipSuccess)Cleanup(false);
displayMatrix1 ("weighted sum",Zh, b, cmdLineArgs.M);
//		 error = hipMemcpy(d_Zh, Zh,b * cmdLineArgs.M * sizeof(double), hipMemcpyHostToDevice);
		            // if (error != hipSuccess) Cleanup(false);
/*	Activation<<<dimGrid, cmdLineArgs.M>>>(d_H,d_Zh);
		error = hipGetLastError();
		             if (error != hipSuccess) Cleanup(false);
					                  hipDeviceSynchronize();
		error = hipMemcpy(H, d_H, b * (cmdLineArgs.M+1) * sizeof(double), hipMemcpyDeviceToHost);
														                      if (error != hipSuccess) Cleanup(false);
		//displayMatrix1 ("activation", H, b, cmdLineArgs.M+1);
*/		mm(Zy,H,Why,b,cmdLineArgs.M+1,cmdLineArgs.P); //Zy=H*Why	

func(P,Zy,b,cmdLineArgs.P,0); //P=fn(Zy)	
/*		 error = hipMemcpy(d_Zy, Zy,b * cmdLineArgs.P * sizeof(double), hipMemcpyHostToDevice);
		                      if (error != hipSuccess) Cleanup(false);
							          Exponents<<<dimGrid, cmdLineArgs.P>>>(d_P,d_Zy);
									          error = hipGetLastError();
											                       if (error != hipSuccess) Cleanup(false);
																hipDeviceSynchronize();
																										         error = hipMemcpy(P, d_P, b * (cmdLineArgs.P) * sizeof(double), hipMemcpyDeviceToHost);                                                                   if (error != hipSuccess) Cleanup(false);
*/		reduction(P,sum,b,cmdLineArgs.P);  //summation of probabilities for each training sample
		prob(P,P1,sum,b,cmdLineArgs.P); //P1=fn(P,sum)	
		//error(E,P1,Y,b,cmdLineArgs.P);	//E=P1-Y
		error = hipMemcpy(d_C, Y, size, hipMemcpyHostToDevice);
		    if (error != hipSuccess) Cleanup(false);
			    error = hipMemcpy(d_B, P1, size, hipMemcpyHostToDevice);
				    if (error != hipSuccess) Cleanup(false);
		 AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
		 error = hipGetLastError();
		     if (error != hipSuccess) Cleanup(false);
			     hipDeviceSynchronize();
				  error = hipMemcpy(E, d_A, size, hipMemcpyDeviceToHost);
				      if (error != hipSuccess) Cleanup(false);

//displayMatrix1 ("expected error", E, b, cmdLineArgs.P);
		/*Backprpagation Phase*/
//		printf("Backward phase\n");
		mtm(dWhy,H,E,cmdLineArgs.M+1,b,cmdLineArgs.P); //dWhy=H'*E ('->transpose)		
		delta(Why,dWhy,cmdLineArgs.M+1,cmdLineArgs.P,learningrate); //Why=fn(dwhy)
		mmt(H,Why,E,b,cmdLineArgs.M+1,cmdLineArgs.P); //H=Why*E'		
		gradient_func(Zh,H,b,cmdLineArgs.M); //Zh=f1"(H) ("->gradient of f1)		
		mtm(dWxh,X,Zh,cmdLineArgs.N+1,b,cmdLineArgs.M);	//dWxh=X'Zh
		delta(Wxh,dWxh,cmdLineArgs.N+1,cmdLineArgs.M,learningrate);//Wxh=fn(dWxh)
	}
	if(k3)
	{
		for(long i=0;i<k3;i++)
		{
		X(i,0)=H(i,0)=1;
	 	memcpy (&X(i,1), &inputs[cmdLineArgs.N*((k2*b)+i)], cmdLineArgs.N*sizeof(double));
		}
		Y = &outputs[k2*b*cmdLineArgs.P];

		/*Forward Phase*/
		mm(Zh,X,Wxh,k3,cmdLineArgs.N+1,cmdLineArgs.M);
		func(H,Zh,k3,cmdLineArgs.M,1);
		mm(Zy,H,Why,k3,cmdLineArgs.M+1,cmdLineArgs.P);		
		func(P,Zy,k3,cmdLineArgs.P,0); 
		reduction(P,sum,k3,cmdLineArgs.P);  
		prob(P,P1,sum,k3,cmdLineArgs.P);  
//		error(E,P1,Y,k3,cmdLineArgs.P);
			
		/*Backprpagation Phase*/ 		
		mtm(dWhy,H,E,cmdLineArgs.M+1,k3,cmdLineArgs.P);
		delta(Why,dWhy,cmdLineArgs.M+1,cmdLineArgs.P,learningrate);
		mmt(H,Why,E,k3,cmdLineArgs.M+1,cmdLineArgs.P);		
		gradient_func(Zh,H,k3,cmdLineArgs.M);		
		mtm(dWxh,X,Zh,cmdLineArgs.N+1,k3,cmdLineArgs.M);
		delta(Wxh,dWxh,cmdLineArgs.N+1,cmdLineArgs.M,learningrate);

	}	
   }

  stop_timer();
  double time = elapsed_time();
  printf( "Time: %lf\n",time);
/*---------------------------------------------------------------------------------------------------------------*/
/*----------------------------------------------Print outputs----------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
   if(cmdLineArgs.V)
   {
	/*Need the following 2 statements for Testing*/
	displayMatrix1 ("input/hidden weights", Wxh, cmdLineArgs.N+1, cmdLineArgs.M);
	displayMatrix1 ("hidden/output weights", Why, cmdLineArgs.M+1, cmdLineArgs.P);
	/* Useful for analyzing the accuracy of prediction */
	/*if(k3)
	{	
		displayVector ("last input", &X[k3-1][1], cmdLineArgs.N);
		displayVector ("last output", Y[k3-1], cmdLineArgs.P);
		displayVector ("predicted output",P1[k3-1], cmdLineArgs.P);
	}
	else
	{
		displayVector ("last input", &X[b-1][1], cmdLineArgs.N);
		displayVector ("last output", Y[b-1], cmdLineArgs.P);
		displayVector ("predicted output",P1[b-1], cmdLineArgs.P);
	}*/
   }
/*---------------------------------------------------------------------------------------------------------------*/
/*----------------------------------------------Free Memory------------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
//free(inputs);
//free(outputs);
//free(X);
//free(Zh);
//free(Zy);
//free(H);
//free(E);
//free(P);
//free(P1);
//free(sum);
//free(Wxh);
//free(Why);
//free(dWxh);
//free(dWhy);
/*-------------------------------------------------------END-----------------------------------------------------*/
return 0;
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
	        
			    // Free device vectors
				    if (d_A)
					        hipFree(d_A);
							    if (d_B)
								        hipFree(d_B);
										    if (d_C)
											        hipFree(d_C);

													    // Free host memory
																						    error = hipDeviceReset();
																										    
																											    if (!noError || error != hipSuccess)
																												        printf("cuda malloc or cuda thread exit failed \n");
																														    
																															    fflush( stdout);
																																    fflush( stderr);

																																	    exit(0);
																																		}
void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
	    {
		      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
			        exit(-1);
					    }                         
						}
