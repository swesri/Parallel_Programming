#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------------------------------------------*/
/// bpl.c
/// For CSU CS475 Fall 2016
/// Instructor: Sanjay Rajopadhye
/// GTA: Swetha Varadarajan
/// Based on code Created by Paul Tero at Existor Ltd as part of a neural networks tutorial
/// Modified by Swetha Varadarajan
/// Created: 2016-11-16
/*---------------------------------------------------------------------------------------------------------------*/

#include <stdio.h> 
#include <stdlib.h> 
#include <string.h> 
#include <math.h> 

#include "timer.h"
#include "util.h"
#include "bunch-new.h"
#include "bplKernel.h"
#include "matmultKernel00.cu"

#define X(i,j) X[((i)*(cmdLineArgs.N+1))+(j)]
#define H(i,j) H[((i)*(cmdLineArgs.M+1))+(j)]

double* d_E; 
double* d_P1; 
double* d_P;
double* d_H;
double* d_Zh;
double* d_Zy;
double* d_X;
double* d_Y;
double* d_Wxh;
double* d_Why;
double* d_dWxh;
double* d_dWhy;
double* d_sum;


// Utility Functions
void Cleanup(bool);
void checkCUDAerrors(const char *msg);

int main(int argc, char** argv) 
{

/*---------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------Command line parsing--------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/

  Params cmdLineArgs;
  parseCmdLineArgs(&cmdLineArgs,argc,argv);

/*---------------------------------------------------------------------------------------------------------------*/
/*-------------------------------------------Variable Declaration------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/

  /*Array description and its size in the comments next to its declation*/

  double *inputs;//Given inputs = total number of samples(S)*number of inputs per sample(N) 
  double *outputs;//Expected outputs = total number of samples(S)*number of outputs per sample(P) 

  double *X;//Input for a given iteration = bunch size(I)*number of inputs per sample(N+1(bias))
  double *Y;//Output for a given iteration = bunch size(I)*number of outputs per sample(P)

  double *Wxh; //Weights in between input and hidden layer = (N+1)*M
  double *Why; //Weights in between input and hidden layer = (M+1)*P
  double *dWxh; //errors Weights in between input and hidden layer = (N+1)*M
  double *dWhy; //errors Weights in between input and hidden layer = (M+1)*P

  double *Zh; //Weighted sum for hidden layer=I*M
  double *H;  // Activation values = I*(M+1)
  double *Zy; //Weighted sum for output layer=I*P 
  double *E;  //Calculated errorss = I*P
  double *P1; //Oredicted output = I*P
  double *P;  // (exp(Zy)) = I*P
  double *sum; //(summation of the P[i]s) = I
  
  double learningrate = 0.0001; /*learning rate */
  long b = cmdLineArgs.sample_per_iter;
  
  long k2 = cmdLineArgs.sample_total/b ; /*number of full bunches */
  long k3 = cmdLineArgs.sample_total-(k2*b); /* size of the partial bunch */
 
/*---------------------------------------------------------------------------------------------------------------*/
/*-------------------------------------------Memory allocations--------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
 
  inputs  = (double*)malloc(cmdLineArgs.sample_total * sizeof(double) * cmdLineArgs.N);
  outputs = (double*)malloc(cmdLineArgs.sample_total * sizeof(double) * cmdLineArgs.P);
  
  sum	  = (double*)malloc((b)*sizeof(double));

  Wxh     = (double*)malloc((cmdLineArgs.N+1) * sizeof(double) *cmdLineArgs.M);
  Why	  = (double*)malloc((cmdLineArgs.M+1) * sizeof(double) *cmdLineArgs.P);
  dWxh    = (double*)malloc((cmdLineArgs.N+1) * sizeof(double) *cmdLineArgs.M);
  dWhy	  = (double*)malloc((cmdLineArgs.M+1) * sizeof(double) *cmdLineArgs.P);

  X	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.N+1));
  E	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));
  P	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));
  P1  	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));
  H	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.M+1));
  Zh  	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.M));
  Zy  	  = (double*)malloc(b*sizeof(double) * (cmdLineArgs.P));

  if( inputs == NULL || outputs == NULL || X == NULL|| H == NULL || dWxh == NULL || dWhy == NULL 
      || Zh == NULL || Zy == NULL || Wxh == NULL || Why == NULL|| E == NULL || P == NULL
	  || P1 == NULL || sum == NULL)
  {
    printf( "Could not allocate memory\n" );
    exit(0);
  }
   size_t size = b * cmdLineArgs.P * sizeof(double);

   hipError_t errors;
   errors = hipMalloc((void**)&d_E, size);
   if (errors != hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_Y, size);
   if (errors != hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_P1, size);
   if (errors != hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_P, size);
   if (errors != hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_Zy, size);
   if (errors != hipSuccess) Cleanup(false);

   errors = hipMalloc((void**)&d_H, b * (cmdLineArgs.M+1) * sizeof(double));
   if (errors != hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_Zh, b * cmdLineArgs.M * sizeof(double));
   if (errors != hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_X, b*(cmdLineArgs.N+1)*sizeof(double));
   if(errors!= hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_sum, b * sizeof(double));
   if (errors != hipSuccess) Cleanup(false);

   errors = hipMalloc((void**)&d_Wxh, cmdLineArgs.M*(cmdLineArgs.N+1)*sizeof(double));
   if(errors!= hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_dWxh, cmdLineArgs.M*(cmdLineArgs.N+1)*sizeof(double));
   if(errors!= hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_Why, cmdLineArgs.P*(cmdLineArgs.M+1)*sizeof(double));
   if(errors!= hipSuccess) Cleanup(false);
   errors = hipMalloc((void**)&d_dWhy, cmdLineArgs.P*(cmdLineArgs.M+1)*sizeof(double));
   if(errors!= hipSuccess) Cleanup(false);

/*---------------------------------------------------------------------------------------------------------------*/
/*----------------------------------------------Initializations--------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/

  initializeW(Wxh,(cmdLineArgs.N+1),cmdLineArgs.M);
  initializeW(Why,(cmdLineArgs.M+1),cmdLineArgs.P);
  initializeI(inputs,cmdLineArgs.sample_total,cmdLineArgs.N);
  initializeO(outputs,cmdLineArgs.sample_total,cmdLineArgs.P);

/*---------------------------------------------------------------------------------------------------------------*/
/*------------------------------------------------Training-------------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
  

dim3 dimGrid(2,2);                    
dim3 dimBlock(2,2);

	errors = hipMemcpy(d_Wxh, Wxh, cmdLineArgs.M*(cmdLineArgs.N+1)*sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_Zh, Zh, b * (cmdLineArgs.M) * sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_H, H, b * (cmdLineArgs.M+1) * sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_Why, Why, cmdLineArgs.P*(cmdLineArgs.M+1)*sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_Zy, Zy, size, hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_P, P, size, hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_P1, P1, size, hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_E, E, size, hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_dWhy, dWhy, cmdLineArgs.P*(cmdLineArgs.M+1)*sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_dWxh, dWxh, cmdLineArgs.M*(cmdLineArgs.N+1)*sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);
	errors = hipMemcpy(d_sum, sum, b*sizeof(double), hipMemcpyHostToDevice);
    	if (errors != hipSuccess) Cleanup(false);

initialize_timer();
start_timer();		

  for (long t=0; t<cmdLineArgs.iter; t++) //Time loop
  {
 	for (long s=0; s<k2; s++) //Bunch loop
	  { 	
		for(long i=0;i<b;i++)
		{
			X(i,0)=H(i,0)=1;//bias setting
			memcpy (&X(i,1), &inputs[cmdLineArgs.N*((s*b)+i)], cmdLineArgs.N*sizeof(double)); 
		}
		Y = &outputs[s*b*cmdLineArgs.P]; 
		errors = hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    		if (errors != hipSuccess) Cleanup(false);
		errors = hipMemcpy(d_X, X, b*(cmdLineArgs.N+1)*sizeof(double), hipMemcpyHostToDevice);
    		if (errors != hipSuccess) Cleanup(false);
		
		//mm(Zh,X,Wxh,b,cmdLineArgs.N+1,cmdLineArgs.M); //Zh=X*Wxh
		MatrixMult<<<dimGrid,dimBlock>>>(d_X,d_Wxh,d_Zh,b,cmdLineArgs.N+1,cmdLineArgs.M,0);
		errors = hipMemcpy(Zh, d_Zh, b * (cmdLineArgs.M)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);
		displayMatrix1 ("weighted sum", Zh, b, cmdLineArgs.M);

		//func(H,Zh,b,cmdLineArgs.M,1);
		//errors = hipMemcpy(d_Zh, Zh, b * (cmdLineArgs.M) * sizeof(double), hipMemcpyHostToDevice);
    	//	if (errors != hipSuccess) Cleanup(false);
		errors = hipMemcpy(d_H, H, b * (cmdLineArgs.M+1) * sizeof(double), hipMemcpyHostToDevice);
		if (errors != hipSuccess) Cleanup(false);
		Activation<<<b, cmdLineArgs.M+1>>>(d_H,d_Zh);
		errors = hipMemcpy(H, d_H, b * (cmdLineArgs.M+1)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);
		mm(Zy,H,Why,b,cmdLineArgs.M+1,cmdLineArgs.P); //Zy=H*Why
		//displayMatrix1("Activation", H, b, cmdLineArgs.M+1);
	
		//func(P,Zy,b,cmdLineArgs.P,0); //P=fn(Zy)
		errors = hipMemcpy(d_Zy, Zy, size, hipMemcpyHostToDevice);
    		if (errors != hipSuccess) Cleanup(false);
		Exponents<<<dimGrid, cmdLineArgs.P>>>(d_P,d_Zy);
		errors = hipMemcpy(P, d_P, b * (cmdLineArgs.P)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);
								                                                              
		//reduction(P,sum,b,cmdLineArgs.P);  
		Reduction<<<b, 1>>>(d_P,d_sum,cmdLineArgs.P);
		errors = hipMemcpy(sum, d_sum, b*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);

		//prob(P,P1,sum,b,cmdLineArgs.P); //P1=fn(P,sum)
		errors = hipMemcpy(d_sum, sum, b*sizeof(double), hipMemcpyHostToDevice);
		if (errors != hipSuccess) Cleanup(false);
		Division<<<b, cmdLineArgs.P>>>(d_P1,d_P,d_sum);
		errors = hipMemcpy(P1, d_P1, b * (cmdLineArgs.P)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);
	
		//error(E,P1,Y,b,cmdLineArgs.P);	//E=P1-Y
		ErrorCalc<<<b, cmdLineArgs.P>>>(d_E,d_P1,d_Y);
		errors = hipMemcpy(E, d_E, size, hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);
		
		//MatrixMult<<<>>>();
		//Activation<<<b, cmdLineArgs.M+1>>>(d_H,d_Zh);
		//MatrixMult<<<>>>();
		//Exponents<<<dimGrid, cmdLineArgs.P>>>(d_P,d_Zy);
		//Reduction<<<b, 1>>>(d_P,d_sum,cmdLineArgs.P);
		//Division<<<b, cmdLineArgs.P>>>(d_P1,d_P,d_sum);
		//ErrorCalc<<<b, cmdLineArgs.P>>>(d_E,d_P1,d_Y);
		
		/*Backprpagation Phase*/
		mtm(dWhy,H,E,cmdLineArgs.M+1,b,cmdLineArgs.P); //dWhy=H'*E ('->transpose)		
		//delta(Why,dWhy,cmdLineArgs.M+1,cmdLineArgs.P,learningrate); //Why=fn(dwhy)
		errors = hipMemcpy(d_Why, Why, cmdLineArgs.P * (cmdLineArgs.M+1) * sizeof(double), hipMemcpyHostToDevice);
    		if (errors != hipSuccess) Cleanup(false);
		errors = hipMemcpy(d_dWhy, dWhy, cmdLineArgs.P * (cmdLineArgs.M+1) * sizeof(double), hipMemcpyHostToDevice);
		if (errors != hipSuccess) Cleanup(false);
		Subtraction<<<cmdLineArgs.P, cmdLineArgs.M+1>>>(d_dWhy,d_Why);
		errors = hipMemcpy(Why, d_Why, cmdLineArgs.P * (cmdLineArgs.M+1)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);

		mmt(H,Why,E,b,cmdLineArgs.M+1,cmdLineArgs.P); //H=Why*E'		
		//gradient_func(Zh,H,b,cmdLineArgs.M); //Zh=f1"(H) ("->gradient of f1)	
		errors = hipMemcpy(d_H, H, b * (cmdLineArgs.M+1) * sizeof(double), hipMemcpyHostToDevice);
		if (errors != hipSuccess) Cleanup(false);
		Gradient<<<b, cmdLineArgs.M>>>(d_Zh,d_H);
		errors = hipMemcpy(Zh, d_Zh, b * (cmdLineArgs.M)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);
		//displayMatrix1(“Gradient”, Zh, b, cmdLineArgs.M);

	
		mtm(dWxh,X,Zh,cmdLineArgs.N+1,b,cmdLineArgs.M);	//dWxh=X'Zh
		//delta(Wxh,dWxh,cmdLineArgs.N+1,cmdLineArgs.M,learningrate);//Wxh=fn(dWxh)
		errors = hipMemcpy(d_Wxh, Wxh, cmdLineArgs.M * (cmdLineArgs.N+1) * sizeof(double), hipMemcpyHostToDevice);
    		if (errors != hipSuccess) Cleanup(false);
		errors = hipMemcpy(d_dWxh, dWxh, cmdLineArgs.M * (cmdLineArgs.N+1) * sizeof(double), hipMemcpyHostToDevice);
		if (errors != hipSuccess) Cleanup(false);
		Subtraction<<<cmdLineArgs.M, cmdLineArgs.N+1>>>(d_dWxh,d_Wxh);
		errors = hipMemcpy(Wxh, d_Wxh, cmdLineArgs.M * (cmdLineArgs.N+1)*sizeof(double), hipMemcpyDeviceToHost);
		if (errors != hipSuccess) Cleanup(false);

		//MatrixMult<<<>>>();
		//Subtraction<<<cmdLineArgs.P, cmdLineArgs.M+1>>>(d_dWhy,d_Why);
		//MatrixMult<<<>>>();
		//Gradient<<<b, cmdLineArgs.M>>>(d_Zh,d_H);
		//MatrixMult<<<>>>();
		//Subtraction<<<cmdLineArgs.M, cmdLineArgs.N+1>>>(d_dWxh,d_Wxh);

	}
	if(k3)
	{
		for(long i=0;i<k3;i++)
		{
		X(i,0)=H(i,0)=1;
	 	memcpy (&X(i,1), &inputs[cmdLineArgs.N*((k2*b)+i)], cmdLineArgs.N*sizeof(double));
		}
		Y = &outputs[k2*b*cmdLineArgs.P];

		/*Forward Phase*/
		mm(Zh,X,Wxh,k3,cmdLineArgs.N+1,cmdLineArgs.M);
		func(H,Zh,k3,cmdLineArgs.M,1);
		mm(Zy,H,Why,k3,cmdLineArgs.M+1,cmdLineArgs.P);		
		func(P,Zy,k3,cmdLineArgs.P,0); 
		reduction(P,sum,k3,cmdLineArgs.P);  
		prob(P,P1,sum,k3,cmdLineArgs.P);  
		error(E,P1,Y,k3,cmdLineArgs.P);
			
		/*Backprpagation Phase*/ 		
		mtm(dWhy,H,E,cmdLineArgs.M+1,k3,cmdLineArgs.P);
		delta(Why,dWhy,cmdLineArgs.M+1,cmdLineArgs.P,learningrate);
		mmt(H,Why,E,k3,cmdLineArgs.M+1,cmdLineArgs.P);		
		gradient_func(Zh,H,k3,cmdLineArgs.M);		
		mtm(dWxh,X,Zh,cmdLineArgs.N+1,k3,cmdLineArgs.M);
		delta(Wxh,dWxh,cmdLineArgs.N+1,cmdLineArgs.M,learningrate);

	}	
   }

  stop_timer();
  double time = elapsed_time();
  printf( "Time: %lf\n",time);

/*error = hipMemcpy(Wxh, d_Wxh, cmdLineArgs.M*(cmdLineArgs.N+1)*sizeof(double), hipMemcpyDeviceToHost);
if (error != hipSuccess) Cleanup(false);
error = hipMemcpy(Why, d_Why, cmdLineArgs.P*(cmdLineArgs.M+1)*sizeof(double), hipMemcpyDeviceToHost);
if (error != hipSuccess) Cleanup(false);*/
	
/*---------------------------------------------------------------------------------------------------------------*/
/*----------------------------------------------Print outputs----------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
   if(cmdLineArgs.V)
   {
	/*Need the following 2 statements for Testing*/
	displayMatrix1 ("input/hidden weights", Wxh, cmdLineArgs.N+1, cmdLineArgs.M);
	displayMatrix1 ("hidden/output weights", Why, cmdLineArgs.M+1, cmdLineArgs.P);
	/* Useful for analyzing the accuracy of prediction */
	/*if(k3)
	{	
		displayVector ("last input", &X[k3-1][1], cmdLineArgs.N);
		displayVector ("last output", Y[k3-1], cmdLineArgs.P);
		displayVector ("predicted output",P1[k3-1], cmdLineArgs.P);
	}
	else
	{
		displayVector ("last input", &X[b-1][1], cmdLineArgs.N);
		displayVector ("last output", Y[b-1], cmdLineArgs.P);
		displayVector ("predicted output",P1[b-1], cmdLineArgs.P);
	}*/
   }
/*---------------------------------------------------------------------------------------------------------------*/
/*----------------------------------------------Free Memory------------------------------------------------------*/
/*---------------------------------------------------------------------------------------------------------------*/
free(inputs);
free(outputs);
free(X);
free(Zh);
free(Zy);
free(H);
free(E);
free(P);
free(P1);
free(sum);
free(Wxh);
free(Why);
free(dWxh);
free(dWhy);
/*-------------------------------------------------------END-----------------------------------------------------*/
return 0;
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_E)
        hipFree(d_E);
    if (d_P)
        hipFree(d_P);
    if (d_P1)
        hipFree(d_P1);

  
        
    error = hipDeviceReset();
    
    if (!noError || error != hipSuccess)
{        printf("cuda malloc or cuda thread exit failed \n");
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}
