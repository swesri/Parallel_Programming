
#include <hip/hip_runtime.h>

//type0:MM,type1:MTM,type2:MMT
#define by 2
#define bx 2
__global__ void MatrixMult(double* A, double* B, double* C, long N, long M, long P, int type){

 
  double *Asub, *Bsub, *Csub;

  int thread_col = threadIdx.y;
  int thread_row = threadIdx.x;
  int block_col = blockIdx.y;
  int block_row = blockIdx.x;

  Csub = &C[P * blockDim.y * block_row + blockDim.x * block_col];

  double Cvalue = 0;

if(type==0)
{
  for (int m = 0;  m < (M / blockDim.x); ++m){
    
    Asub = &A[M * blockDim.y * block_row + blockDim.x * m];
    Bsub = &B[M * blockDim.x * block_col + blockDim.y * m];


    __shared__ double shared_A[by][bx];
    __shared__ double shared_B[bx][by];

  
    shared_A[thread_row][thread_col] = Asub[thread_row * M/*blockDim.x*/ + thread_col];
    shared_B[thread_row][thread_col] = Bsub[thread_row * M/*blockDim.y*/ + thread_col];

   
    __syncthreads();

    
#pragma unroll
    for(int e=0; e<blockDim.x; ++e)
       Cvalue += shared_A[thread_row][e] * shared_B[e][thread_col];

    __syncthreads();
  }
}

//printf("(%d,%d)=%d\n",blockIdx.x,blockIdx.y,thread_row*blockDim.x+thread_col);
//printf("ThreadIds=%d,%d\n",threadIdx.x,threadIdx.y);
//printf("global=%d\tlocal=%d\tvalue=%lf\n",(P * blockDim.y * block_row + blockDim.x * block_col),(thread_row * blockDim.x + thread_col),Cvalue);
Csub[thread_row * P/*blockDim.x*/ + thread_col] = Cvalue;
}

